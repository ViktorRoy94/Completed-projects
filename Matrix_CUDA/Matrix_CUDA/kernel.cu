#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.cuh"
__global__ void kernelMatrixMul(int* A, int* B, int* C, int n)
{
	int i = threadIdx.x;
	int j = blockIdx.x;
	for(int k=0; k<n; k++)
	{
		C[i*n+j]+=A[i*n+k]*B[k*n+j];
	}
}

void StartCuda(int* A, int* B, int* C, int n)
{
	kernelMatrixMul<<<n, n>>>(A,B,C,n);
	hipDeviceSynchronize();
}

