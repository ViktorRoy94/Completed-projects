#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <math.h>
#include <stdio.h>

__device__ int flag;														

 __global__ void methodJacobi(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int N, int *iteration, double dt, double *f) // eps - òî÷íîñòü âû÷èñëåíèé, äî êîòîðûõ âûïîëíÿåò ìåòîä
	{
		double *tmp;
		int iter = 0;														// ê-âî èòåðàöèé â êàæäîì ïîòîêå
		int index; 															// èíäåêñ ïîòîêà
		do{
			index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - íîìåð ïîòîêà, blockIdx.x - áëîêà
			if( index == 0 ){												// íóëåâîé ïîòîê óñòàíàâëèâàåò flag â 0
				flag = 0;
			}
			__syncthreads();
			while (index < N)
			{
				x2[index] = 0;
				for (int j=rowIndex[index]; j<rowIndex[index+1]; j++)		// óìíîæåíèå crs ìàòðèöû íà âåêòîð
					x2[index] +=value[j] * x1[col[j]];				
				x2[index] =  x1[index] + ( -x2[index] + b[index] ) / diag;  // ðàññ÷åò ñëåäóþùåãî çíà÷åíèÿ íà îñíîâå ïðåäûäóùåãî
				
				if (fabs(x1[index]-x2[index]) > eps){
					flag = index+1;  
				}
			 	index += blockDim.x * gridDim.x;						 // gridDim.x - ðàçìåð ñåòêè â áëîêàõ, blockDim.x - ðàçìåðû áëîêà â ïîòîêàõ
			 }
			tmp = x1;													//			
			x1 = x2;													// ìåíÿåì ïðåäûäóùèÿ è ñëåäóþùèå çíà÷åíèÿ ìåñòàìè
			x2 = tmp;													//
			iter++;														// óâåëè÷èâàåì èòåðàöèè		
			__syncthreads();

		}while ( flag  );												// åñëè êòî-òî çàïèñàë âî ôëàã -> âûïîëíÿåì äàëüøå
		
		index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - íîìåð ïîòîêà, blockIdx.x - áëîêà
		while (index < N){
			b[index]=f[index]*dt+x1[index];
			index += blockDim.x * gridDim.x;						 // gridDim.x - ðàçìåð ñåòêè â áëîêàõ, blockDim.x - ðàçìåðû áëîêà â ïîòîêàõ
		}

		if( threadIdx.x + blockIdx.x * blockDim.x == 0 ){
			*iteration = iter; 
		}

	}

void StartCuda(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int *iteration, int Sx, int Sy, int Sz, double dt, double *f)
{
	methodJacobi<<<1,512>>>(value, col, rowIndex, x1, x2, b, eps, diag, (Sx-2)*(Sy-2)*(Sz-2), iteration,dt,f); 
	hipDeviceSynchronize();
	//hipError_t err;
	//err = hipGetLastError();
	//printf("err=%d, %s\n", (int)err, hipGetErrorString(err));
}