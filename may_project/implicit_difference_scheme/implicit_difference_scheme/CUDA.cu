#include "hip/hip_runtime.h"
﻿#include "CUDA.cuh"
#include <math.h>
#include <stdio.h>

__device__ int flag;														

 __global__ void methodJacobi(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int N, int *iteration, double dt, double *f) // eps - ц╡ц╝ц╥ц╜ц╝ц╠ц╡ц╪ ц╒ц╩ц╥ц╗ц╠ц╚ц╔ц╜ц╗ц╘, ц╓ц╝ ц╙ц╝ц╡ц╝ц╟ц╩ц╣ ц╒ц╩ц╞ц╝ц╚ц╜ц©ц╔ц╡ ц╛ц╔ц╡ц╝ц╓
	{
		double *tmp;
		int iter = 0;														// ц╙-ц╒ц╝ ц╗ц╡ц╔ц╟ц═ц╤ц╗ц╘ ц╒ ц╙ц═ц╕ц╓ц╝ц╛ ц╞ц╝ц╡ц╝ц╙ц╔
		int index; 															// ц╗ц╜ц╓ц╔ц╙ц╠ ц╞ц╝ц╡ц╝ц╙ц═
		do{
			index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - ц╜ц╝ц╛ц╔ц╟ ц╞ц╝ц╡ц╝ц╙ц═, blockIdx.x - ц║ц╚ц╝ц╙ц═
			if( index == 0 ){												// ц╜цЁц╚ц╔ц╒ц╝ц╘ ц╞ц╝ц╡ц╝ц╙ цЁц╠ц╡ц═ц╜ц═ц╒ц╚ц╗ц╒ц═ц╔ц╡ flag ц╒ 0
				flag = 0;
			}
			__syncthreads();
			while (index < N)
			{
				x2[index] = 0;
				for (int j = rowIndex[index]; j < rowIndex[index+1]; j++)		// цЁц╛ц╜ц╝ц╕ц╔ц╜ц╗ц╔ crs ц╛ц═ц╡ц╟ц╗ц╤ц╩ ц╜ц═ ц╒ц╔ц╙ц╡ц╝ц╟
					x2[index] += value[j] * x1[col[j]];				
				x2[index] =  x1[index] + ( -x2[index] + b[index] ) / diag;  // ц╟ц═ц╠ц╠ц╥ц╔ц╡ ц╠ц╚ц╔ц╓цЁц╬ц╧ц╔цёц╝ ц╖ц╜ц═ц╥ц╔ц╜ц╗ц© ц╜ц═ ц╝ц╠ц╜ц╝ц╒ц╔ ц╞ц╟ц╔ц╓ц╩ц╓цЁц╧ц╔цёц╝
				
				if (fabs(x1[index]-x2[index]) > eps){
					flag = index+1;  
				}
			 	index += blockDim.x * gridDim.x;						 // gridDim.x - ц╟ц═ц╖ц╛ц╔ц╟ ц╠ц╔ц╡ц╙ц╗ ц╒ ц║ц╚ц╝ц╙ц═ц╣, blockDim.x - ц╟ц═ц╖ц╛ц╔ц╟ц╩ ц║ц╚ц╝ц╙ц═ ц╒ ц╞ц╝ц╡ц╝ц╙ц═ц╣
			 }
			tmp = x1;													//			
			x1 = x2;													// ц╛ц╔ц╜ц©ц╔ц╛ ц╞ц╟ц╔ц╓ц╩ц╓цЁц╧ц╗ц© ц╗ ц╠ц╚ц╔ц╓цЁц╬ц╧ц╗ц╔ ц╖ц╜ц═ц╥ц╔ц╜ц╗ц© ц╛ц╔ц╠ц╡ц═ц╛ц╗
			x2 = tmp;													//
			iter++;														// цЁц╒ц╔ц╚ц╗ц╥ц╗ц╒ц═ц╔ц╛ ц╗ц╡ц╔ц╟ц═ц╤ц╗ц╗		
			__syncthreads();

		}while ( flag  );												// ц╔ц╠ц╚ц╗ ц╙ц╡ц╝-ц╡ц╝ ц╖ц═ц╞ц╗ц╠ц═ц╚ ц╒ц╝ ц╢ц╚ц═цё -> ц╒ц╩ц╞ц╝ц╚ц╜ц©ц╔ц╛ ц╓ц═ц╚ц╪ц╦ц╔
		
		index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - ц╜ц╝ц╛ц╔ц╟ ц╞ц╝ц╡ц╝ц╙ц═, blockIdx.x - ц║ц╚ц╝ц╙ц═
		while (index < N){
			b[index]=f[index]*dt+x1[index];
			index += blockDim.x * gridDim.x;						 // gridDim.x - ц╟ц═ц╖ц╛ц╔ц╟ ц╠ц╔ц╡ц╙ц╗ ц╒ ц║ц╚ц╝ц╙ц═ц╣, blockDim.x - ц╟ц═ц╖ц╛ц╔ц╟ц╩ ц║ц╚ц╝ц╙ц═ ц╒ ц╞ц╝ц╡ц╝ц╙ц═ц╣
		}

		if( threadIdx.x + blockIdx.x * blockDim.x == 0 ){
			*iteration = iter; 
		}

	}

void StartCuda(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int *iteration, int Sx, int Sy, int Sz, double dt, double *f)
{
	methodJacobi<<<1,512>>>(value, col, rowIndex, x1, x2, b, eps, diag, (Sx-2)*(Sy-2)*(Sz-2), iteration,dt,f); 
	hipDeviceSynchronize();
	//hipError_t err;
	//err = hipGetLastError();
	//printf("err=%d, %s\n", (int)err, hipGetErrorString(err));
}