#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <math.h>
#include <stdio.h>

__device__ int flag;														

 __global__ void methodJacobi(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int N, int *iteration, double dt, double *f) // eps - �������� ����������, �� ������� ��������� �����
	{
		double *tmp;
		int iter = 0;														// �-�� �������� � ������ ������
		int index; 															// ������ ������
		do{
			index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - ����� ������, blockIdx.x - �����
			if( index == 0 ){												// ������� ����� ������������� flag � 0
				flag = 0;
			}
			__syncthreads();
			while (index < N)
			{
				x2[index] = 0;
				for (int j=rowIndex[index]; j<rowIndex[index+1]; j++)		// ��������� crs ������� �� ������
					x2[index] +=value[j] * x1[col[j]];				
				x2[index] =  x1[index] + ( -x2[index] + b[index] ) / diag;  // ������� ���������� �������� �� ������ �����������
				
				if (fabs(x1[index]-x2[index]) > eps){
					flag = index+1;  
				}
			 	index += blockDim.x * gridDim.x;						 // gridDim.x - ������ ����� � ������, blockDim.x - ������� ����� � �������
			 }
			tmp = x1;													//			
			x1 = x2;													// ������ ���������� � ��������� �������� �������
			x2 = tmp;													//
			iter++;														// ����������� ��������		
			__syncthreads();

		}while ( flag  );												// ���� ���-�� ������� �� ���� -> ��������� ������
		
		index = threadIdx.x + blockIdx.x * blockDim.x;					// threadIdx.x - ����� ������, blockIdx.x - �����
		while (index < N){
			b[index]=f[index]*dt+x1[index];
			index += blockDim.x * gridDim.x;						 // gridDim.x - ������ ����� � ������, blockDim.x - ������� ����� � �������
		}

		if( threadIdx.x + blockIdx.x * blockDim.x == 0 ){
			*iteration = iter; 
		}

	}

void StartCuda(double* value, int* col, int* rowIndex, double* x1, double* x2, double *b, double eps, double diag, int *iteration, int Sx, int Sy, int Sz, double dt, double *f)
{
	methodJacobi<<<1,512>>>(value, col, rowIndex, x1, x2, b, eps, diag, (Sx-2)*(Sy-2)*(Sz-2), iteration,dt,f); 
	hipDeviceSynchronize();
	//hipError_t err;
	//err = hipGetLastError();
	//printf("err=%d, %s\n", (int)err, hipGetErrorString(err));
}