#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void kernel(int *image, int *bicub, int N, int bicImgRows, int bicImCols, int imgRows, int imgCols)
{
	int j = 1 + threadIdx.x + blockIdx.x * blockDim.x;
	int i = 1 + threadIdx.y + blockIdx.y * blockDim.y;
	/*int i = threadIdx.x;
	int j = blockIdx.y;*/
	//printf("In kernel");
	//for (int j = 1; j < bicImgRows-1; j++) 
	while (j < bicImgRows - 2)
	{
		float tmp = (float) (j) / (float) (bicImgRows - 1) * (imgRows - 1);
		int h = (int) floor(tmp);
		if (h < 1) 
		{
			h = 1;
		} 
		else 
		{
			if (h >= imgRows - 2) 
			{
				h = imgRows - 3;
			}
		}
		float u = tmp - h;
		//for (int i = 1; i < bicImCols-1; i++) 
		while ( i < bicImCols - 2)
		{
			tmp = (float) (i) / (float) (bicImCols - 1) * (imgCols - 1);
			int w = (int) floor(tmp);
			if (w < 1) 
			{
				w = 1;
			} 
			else 
			{
				if (w >= imgCols - 2) 
				{
					w = imgCols - 3;
				}
			}
			float t = tmp - w;

		    // ������������  

			float b1 =  (1.0f/4)*(u - 1)*(u - 2)*(u + 1)*(t - 1)*(t - 2)*(t + 1);
			float b2 = -  (1.0f/4) *(u)*  (u + 1)*(u - 2)*(t - 1)*(t - 2)*(t + 1)  ;
			float b3 = -  (1.0f/4)*(t)*(u - 1)*(u - 2)*(u + 1)*(t + 1)*(t - 2)  ;
			float b4 =   (1.0f/4)*(u)*(t)*(u + 1)*(u - 2)*(t + 1)*(t - 2)  ;
			float b5 = -  (1.0f/12)*(u)*(u - 1)*(u - 2)*(t - 1)*(t - 2)*(t + 1)  ;
			float b6 = -  (1.0f/12)*(t)*(u - 1)*(u - 2)*(u + 1)*(t - 1)*(t - 2)  ;
			float b7 =   (1.0f/12)*(u)*(t)*(u - 1)*(u - 2)*(t + 1)*(t - 2)  ;
			float b8 =   (1.0f/12)*(u)*(t)*(u + 1)*(u - 2)*(t - 1)*(t - 2)  ;
			float b9 =   (1.0f/12)*(u)*(u - 1)*(u + 1)*(t - 1)*(t - 2)*(t + 1)  ;
			float b10 =   (1.0f/12)*(t)*(u - 1)*(u - 2)*(u + 1)*(t - 1)*(t + 1)  ;
			float b11 =   (1.0f/36)*(u)*(t)*(u - 1)*(u - 2)*(t - 1)*(t - 2)  ;
			float b12 = -  (1.0f/12)*(u)*(t)*(u - 1)*(u + 1)*(t + 1)*(t - 2)  ;
			float b13 = -  (1.0f/12)*(u)*(t)*(u + 1)*(u - 2)*(t - 1)*(t + 1)  ;
			float b14 =  -  (1.0f/36)*(u)*(t)*(u - 1)*(u + 1)*(t - 1)*(t - 2)  ;
			float b15 =  -  (1.0f/36)*(u)*(t)*(u - 1)*(u - 2)*(t - 1)*(t + 1)  ;
			float b16 =   (1.0f/36)*(u)*(t)*(u - 1)*(u + 1)*(t - 1)*(t + 1) ;


			// ��������� �������: a[i][j] 
			float p1 = image[h * imgCols + w];
			float p2 = image[(h+1) * imgCols + w];
			float p3 = image[h * imgCols + w+1];
			float p4 = image[(h+1) * imgCols + w+1];
			float p5 = image[(h-1) * imgCols + w];
			float p6 = image[h * imgCols + w-1];
			float p7 = image[(h-1) * imgCols + w+1];
			float p8 = image[(h+1) * imgCols + w-1];
			float p9 = image[(h+2) * imgCols + w];
			float p10 = image[(h) * imgCols + w+2];
			float p11= image[(h-1) * imgCols + w-1];
			float p12= image[(h+2) * imgCols + w+1];
			float p13= image[(h+1) * imgCols + w+2];
			float p14= image[(h+2) * imgCols + w-1];
			float p15= image[(h-1) * imgCols + w+2];
			float p16= image[(h+2) * imgCols + w+2];


			// ���������� 
			float newPixel = p1 *b1 + p2 *b2 + p3 *b3 +  p4 *b4 +  p5 *b5+  p6 *b6+  p7 *b7+  p8 *b8+  p9 *b9+  p10 *b10+  p11 *b11+  p12 *b12+  p13 *b13+  p14 *b14+  p15 *b15+  p16 *b16;

			// ����� ������ 
			bicub[j *bicImCols + i] = abs(newPixel);
			i += blockDim.y * gridDim.y;
		}
		j += blockDim.x * gridDim.x;
	}
}

void StartCuda(int *image, int *bicub, int N, int bicImgRows, int bicImCols, int imgRows, int imgCols)
{
	dim3 block( (bicImgRows + 511) / 512, (bicImCols + 511) / 512);
	dim3 grid(512, 512);
	//dim3 grid(2560, 2560);
	//dim3 block(16, 16);
	printf("\nstart kernel");
	kernel<<<grid, block>>>(image, bicub, N, bicImgRows, bicImCols, imgRows, imgCols);
	//kernel<<<1024, 1024>>>(image, bicub, N, bicImgRows, bicImCols, imgRows, imgCols);
}
