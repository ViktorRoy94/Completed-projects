#include "hip/hip_runtime.h"
#include "CUDA.cuh"
 
 int flag = 0; // ���� � flag, ���� ���� �����, ������� 1 => �������� ����� x1 � �2 > eps
 int iteration = 1;						// ������� �������� ������ ��� ���������� �������     

 __global__ int methodJacobi(crsMatrix A, double* x1, double* x2, double *b, double eps,double diag) // eps - �������� ����������, �� ������� ��������� �����
	{
		
		double * Multip = A.Multiplicate(x1);	// �������� ������� � �� ������ �1
		for ( int i=0; i<N; i++)
		{
			x2[i] = x1[i] - (Multip[i] - b[i])/diag;
		}
		while (flag==1)
		{
			flag = 0;
			if (fabs(x1[i]-x2[i]) > eps) flag++;  // x1[i],x2[i] �������� �� x1[�����1] � �.�.
			
			delete(Multip);
			for(unsigned int i=0; i<N; i++)
			{
				x1[i]=x2[i];
			}
			Multip = A.Multiplicate(x2);
			for ( int i=0; i<N; i++)	
			{
				x2[i] = x2[i] - (Multip[i] - b[i])/diag;
			}
			iteration++;
		}
		return iteration;
	}


	/*double dev(double* x1, double* x2)
		{
			double max = fabs(x1[0]-x2[0]);
			double tmp;
			for (int i=1; i<N; i++)
			{
				tmp = fabs(x1[i]-x2[i]);
				if (max > tmp) max = tmp;
			}
			return max;
		}*/

void StartCuda(crsMatrix A, double* x1, double* x2, double *b, double eps, double diag)
{
	methodJacobi<<<Sy-2,Sz-2>>>(A, x1, x2, b, eps, diag);
	hipDeviceSynchronize();
}