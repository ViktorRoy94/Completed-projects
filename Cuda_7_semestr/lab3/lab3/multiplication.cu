#include "hip/hip_runtime.h"
#include "multiplication.h"

__global__ void kernel (const int * a, const int * b, int * c, int m, int n, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	c[i*k + j] = 0;
	if(i < m && j < k)
    for (int k_ = 0; k_ < n; ++k_)
		c[i*k + j] += a[i*n + k_] * b[k_*k + j]; 
}


void multiplication_gpu(const int * a, const int * b, int * c, int m, int n, int k)
{
	dim3 numBlocks(m/BLOCK_SIZE, k/BLOCK_SIZE);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

	kernel<<<numBlocks,blockSize>>>(a,  b, c, m, n,  k);
	
}

__global__ void kernel1 (const int * a, const int * b, int * c, int m, int n, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ int partA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int partB[BLOCK_SIZE][BLOCK_SIZE];
	int result = 0;
	for(int k_ = 0; k_ < n/BLOCK_SIZE; k_++)
	{
		partA[threadIdx.x][threadIdx.y] = a[i*n + BLOCK_SIZE*k_ + threadIdx.x];
		partB[threadIdx.x][threadIdx.y] = b[(threadIdx.y + BLOCK_SIZE*k_)*n + j];
		__syncthreads();
		for (int q = 0; q < BLOCK_SIZE; ++q)
			result += partA[q][threadIdx.y] * partB[threadIdx.x][q];
		__syncthreads();
	}
	c[i*k+j] = result;
}

void multiplication_gpu_opt(const int * a, const int * b, int * c, int m, int n, int k)
{
	dim3 numBlocks(k/BLOCK_SIZE, m/BLOCK_SIZE);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	kernel1<<<numBlocks,blockSize>>>(a, b, c, m, n, k);
}
