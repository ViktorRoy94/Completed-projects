
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <string>

#include <hipblas.h>
#include <hiprand/hiprand.h>

#define NONE_TRANS 'N'
#define BLOCK_SIZE 32

__global__ void makePositiveOrientire( const int n, float* matrix )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i < n )
    {
        matrix[ i * n + i ] += n;
    }
}

int main( int argc, char** argv )
{
    const int    n       = 100;
    const int    nIters  = 1000;
    const float  epsilon = 0.0001;

    cublasInit();
    hiprandGenerator_t generator;
    hiprandCreateGenerator( &generator, hiprandRngType_t::HIPRAND_RNG_PSEUDO_DEFAULT );
    hiprandSetPseudoRandomGeneratorSeed( generator, 2671ULL );

    float* devA = nullptr;
    float* devX = nullptr;
    float* devB = nullptr;

    cublasAlloc( n*n, sizeof( float ), (void**) &devA );
    cublasAlloc( n*1, sizeof( float ), (void**) &devX );
    cublasAlloc( n*1, sizeof( float ), (void**) &devB );

    hiprandGenerateUniform( generator, devA, n*n );
    hiprandGenerateUniform( generator, devB, n*1   );

    makePositiveOrientire<<< ( n + BLOCK_SIZE - 1 ) / BLOCK_SIZE, BLOCK_SIZE >>>( n, devA );
    hipDeviceSynchronize();

    hipMemcpy((void*) devX, (const void*) devB, sizeof( float )*n, hipMemcpyDeviceToDevice);

    int    iter  = 0;
    float  error = 0.0f;
    float* devR  = nullptr;
    float* devE  = nullptr;

    cublasAlloc( n*1, sizeof( float ), (void**) &devR );
    cublasAlloc( n*1, sizeof( float ), (void**) &devE );

    do
    {
        /* r = A*x */
        cublasSgemv(
            NONE_TRANS, n, n,
            1.0f, (const float*) devA, n, (const float*) devX, 1, 0.0f, devR, 1
        );

        /* r = r - b */
        cublasSaxpy( n, -1.0f, (const float*) devB, 1, devR, 1 );

        /* e = A*r */
        cublasSgemv(
            NONE_TRANS, n, n,
            1.0f, (const float*) devA, n, (const float*) devR, 1, 0.0f, devE, 1
        );

        float t  = cublasSdot( n, (const float*) devE, 1, (const float*) devR, 1 );
              t /= cublasSdot( n, (const float*) devE, 1, (const float*) devE, 1 );
        
        /* x = x - t * r */
        cublasSaxpy( n, -t, (const float*) devR, 1, devX, 1 );

        /* e = A*x */
        cublasSgemv(
            NONE_TRANS, n, n,
            1.0f, (const float*) devA, n, (const float*) devX, 1, 0.0f, devE, 1
        );

        /* e = e - b */
        cublasSaxpy( n, -1, (const float*) devB, 1, devE, 1 );

        /* error = ||e|| */
        error = cublasSnrm2( n, (const float*) devE, 1 );
    } while ( nIters > iter && error > epsilon);

    std::cout << "diff = " << error << std::endl;
	system("pause");
    cublasFree( devE );
    cublasFree( devR );

    cublasFree( devA );
    cublasFree( devX );
    cublasFree( devB );

    cublasShutdown();
    hiprandDestroyGenerator( generator );

	
    return 0;
}
