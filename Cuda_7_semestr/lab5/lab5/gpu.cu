#include "hip/hip_runtime.h"
#include "convolution.h"


__global__ void gpu(hipfftComplex * a, hipfftComplex * b, int n, hipfftComplex * result)
{
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < n)
	{
		result[i].x = (a[i].x*b[i].x-a[i].y*b[i].y) / n;
		result[i].y = (a[i].x*b[i].y+a[i].y*b[i].x) / n;
	}
}

void convolve_gpu(hipfftComplex * a, hipfftComplex * b, int n, hipfftComplex * result)
{
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, a, a, HIPFFT_FORWARD);
	hipfftExecC2C(plan, b, b, HIPFFT_FORWARD);
	dim3 numBlocks((n-1)/256+1);
	dim3 blockSize(256);
	gpu<<<numBlocks, blockSize>>>(a,  b,  n, result);

	hipfftExecC2C(plan, result, result, HIPFFT_BACKWARD);

	hipfftDestroy(plan);

}
