
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void kernel(int* a, int n)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if ( tid < n) {
		a[tid] = a[tid] + tid;
		printf("a[i] = %d\n", a[tid]);
	}
}

int main()
{
	const int n = 100;
	int a[n];
	int *dev_a;
	
	// started values
	for (int i = 0; i < n; i++)
	{
		a[i] = i;
	}

	// memory on GPU
	hipMalloc((void**)&dev_a, n * sizeof(int));

	// copy to GPU
	hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    kernel<<<10, 10>>>(dev_a, n);
    
	hipMemcpy(a, dev_a, n * sizeof(int), hipMemcpyDeviceToHost);
	
	// output
	for( int i = 0; i < n; i++) {
		cout << "a[i] = " << a[i] << endl;
	}
	
	// release memory
	hipFree(dev_a);

	system("Pause");
    return 0;
}

