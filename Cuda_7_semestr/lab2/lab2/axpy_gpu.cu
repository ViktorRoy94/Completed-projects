#include "hip/hip_runtime.h"
#include "axpy.h"

__global__ void summ_float(int n, float a, float * x, int incx, float * y, int incy)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		for (int j = 0 ; j < 100; ++j)
			y[i * incy] += a * x[i * incx] * cos((float)j);	
	}
}

void saxpy_gpu(int n, float a, float * x, int incx, float * y, int incy)
{
	const int block_size = 128;
	int num_blocks = (n-1)/block_size + 1;
	summ_float<<<num_blocks, block_size>>>(n, a, x, incx, y, incy);
}


__global__ void summ_double(int n, double a, double * x, int incx, double * y, int incy)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		 y[i * incy] += a * x[i * incx];
}


void daxpy_gpu(int n, double a, double * x, int incx, double * y, int incy)
{
	const int block_size = 128;
	int num_blocks = (n-1)/block_size + 1;
	summ_double<<<num_blocks, block_size>>>(n, a, x, incx, y, incy);
}