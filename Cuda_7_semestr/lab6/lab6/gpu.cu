#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <cmath>

#include <hip/hip_runtime_api.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "gpu.h"

#define SIZE_BLOCK 128

__global__ void Check(int n, float *x, float *y, bool *count, int a, int b)
{
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

	x[globalIndex] = a + x[globalIndex] * (b - a);
	if (globalIndex < n)
	{
		if (y[globalIndex] <= 1 / (1 + (x[globalIndex]) * (x[globalIndex]))) 
			count[globalIndex] = 1;
		else 
			count[globalIndex] = 0;
	}
}
__global__ void initCurand(hiprandStateXORWOW_t *state, int n)
{
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (globalIndex < n) 
		hiprand_init(3598, globalIndex, 0, &state[globalIndex]);
}
__global__ void monteCarlo(hiprandStateXORWOW_t *state, int n, int a, int b, bool *res)
{
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (globalIndex < n)
	{
		hiprandStateXORWOW_t localState = state[globalIndex];

		float x =  a + hiprand_uniform(&localState) * (b - a);
		float y = hiprand_uniform(&localState);

		if (y <= 1 / (1 + (x * x))) 
			res[globalIndex] = 1;
		else 
			res[globalIndex] = 0;

		state[globalIndex] = localState;
	}
}
void host(int n, float *x, float *y, bool *count, int a, int b)
{
	dim3 NUM_BLOCKS((n - 1) / SIZE_BLOCK + 1);
	dim3 BLOCK_SIZE(SIZE_BLOCK);
	Check << <NUM_BLOCKS, BLOCK_SIZE >> >(n, x, y, count, a, b);
	hipDeviceSynchronize();
}
void gpu(int n, bool *count, int a, int b)
{
	dim3 NUM_BLOCKS((n - 1) / SIZE_BLOCK + 1);
	dim3 BLOCK_SIZE(SIZE_BLOCK);

	hiprandStateXORWOW_t * devStates;
	hipMalloc ((void**)&devStates , n * sizeof(hiprandStateXORWOW_t));

	initCurand   <<<NUM_BLOCKS, BLOCK_SIZE >> >(devStates, n);
	monteCarlo   <<<NUM_BLOCKS, BLOCK_SIZE >> >(devStates, n, a, b, count);

	hipDeviceSynchronize();
}