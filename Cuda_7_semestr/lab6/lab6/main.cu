#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <>
#include <hiprand.h>
#include <hipfft/hipfft.h>

#include "gpu.h"

#define n 100000 
#define a -4
#define b  4 

int main(int argc, char** argv)
{
	float integral = (float)atan(b) - (float)atan(a);

	hiprandGenerator_t curandGenerator;
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 8694ULL);

	float *x, *y;
	bool *dev_count, *count;

	count  = (bool*)calloc(n, sizeof(bool));

	hipMalloc((void**)&x,         n * sizeof(float));
	hipMalloc((void**)&y,         n * sizeof(float));
	hipMalloc((void**)&dev_count,  n * sizeof(bool));

	hiprandGenerateUniform(curandGenerator, x, n);
	hiprandGenerateUniform(curandGenerator, y, n);

	hiprandDestroyGenerator(curandGenerator);

	host(n, x, y, dev_count, a, b);

	hipMemcpy(count, dev_count, n* sizeof(bool), hipMemcpyDeviceToHost);

	int k = 0;
	for (int i = 0; i < n; i++) 
		if (count[i] == 1) k++;
	float hostIntegral = (float)(abs(b - a) * k) / n;
	
	gpu(n, dev_count, a, b);

	hipMemcpy(count, dev_count, n * sizeof(bool), hipMemcpyDeviceToHost);

	k = 0;
	for (int i = 0; i < n; i++) 
		if (count[i] == 1) k++;
	float monteCarloIntegral = (float)(abs(b - a) * k) / n;

	printf("default     %f\nhost        %f\nMonte Carlo %f\n", integral, hostIntegral, monteCarloIntegral);

	system("pause");
	return 0;
}



