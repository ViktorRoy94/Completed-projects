#include "hip/hip_runtime.h"
﻿
#include "kernel.cuh"

__global__ void kernel(int *image, int *bilin, int N, int bilImgRows, int bilImCols, int imgRows, int imgCols)
{
	int j = 1 + threadIdx.x + blockIdx.x * blockDim.x;
	int i = 1 + threadIdx.y + blockIdx.y * blockDim.y;
	int h, w;
    float t;
    float u;
    float tmp;
    float d1, d2, d3, d4;
    int p1, p2, p3, p4;	/* Окрестные пикселы */
	while (j < bilImgRows - 2)
	{
		tmp = (float) (j) / (float) (bilImgRows-1) * (imgRows-1);
		h = (int) floor(tmp);
		if (h < 0) { h = 0; }
		else
		{
			if (h >= imgRows - 1) {
				h = imgRows - 2;
			}
		}
		u = tmp - h;
		while ( i < bilImCols - 2)
		{
			tmp = (float) (i) / (float) (bilImCols-1) * (imgCols-1);
			w = (int) floor(tmp);
			if (w < 0) { w = 0; }
			else
			{
				if (w >= imgCols - 1) {
					w = imgCols - 2;
				}
			}
			t = tmp - w;
	
			d1 = (1 - t) * (1 - u);
			d2 = t * (1 - u);
			d3 = t * u;
			d4 = (1 - t) * u;
			
			/* Окрестные пиксели: image.at<uchar>(i,j) */
			p1 = image[h * imgCols + w];
			p2 = image[h * imgCols + w+1];
			p3 = image[(h+1) * imgCols + w+1];
			p4 = image[(h+1) * imgCols + w];

			float pixel = p1*d1 + p2*d2 + p3*d3 + p4*d4;
			bilin[j * bilImCols + i] = pixel;
			i += blockDim.y * gridDim.y;
		}
		j += blockDim.x * gridDim.x;
	}
}

void StartCuda(int *image, int *bilin, int N, int bilImgRows, int bilImgCols, int imgRows, int imgCols)
{
	dim3 block( (bilImgRows + 511) / 512, (bilImgCols + 511) / 512);
	dim3 grid(512, 512);
	printf("\nstart kernel");
	kernel<<<grid, block>>>(image, bilin, N, bilImgRows, bilImgCols, imgRows, imgCols);
}
