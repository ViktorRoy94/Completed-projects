#include "hip/hip_runtime.h"
#include <stdio.h>
#include "CUDA.cuh"
__device__ double function(double x, double y, double z)
{
    double fxyz=0.0;
    if (((x>=5.0) && (x<=5.2)) && ((y>=5.0) && (y<=5.2)) && ((z>=5.0) && (z<=5.2)))
        fxyz=4.0;
    return fxyz;
}

__global__ void kernelMatrix(double* masprev, double* masnext,int Sx, int Sy, int Sz,double dx, double dy, double dz,double x0,double y0,double z0,double dt,double ddx,double ddy,double ddz,double *f)
{
	int x = threadIdx.x+1;
	int z = blockIdx.x+1;
    for(int y=1; y<Sx-1; y++)
    {
		masnext[x+y*Sx+z*Sx*Sy]=
			dt*((masprev[(x+1)+y*Sx+z*Sx*Sy]-2*masprev[x+y*Sx+z*Sx*Sy]+masprev[(x-1)+y*Sx+z*Sx*Sy])*(ddx)
			+(masprev[x+(y+1)*Sx+z*Sx*Sy]-2*masprev[x+y*Sx+z*Sx*Sy]+masprev[x+(y-1)*Sx+z*Sx*Sy])*(ddy)
			+(masprev[x+y*Sx+(z+1)*Sx*Sy]-2*masprev[x+y*Sx+z*Sx*Sy]+masprev[x+y*Sx+(z-1)*Sx*Sy])*(ddz)
			+f[x+y*Sx+z*Sx*Sy]-masprev[x+y*Sx+z*Sx*Sy])+masprev[x+y*Sx+z*Sx*Sy];
    }
}


void StartCuda(double* masprev, double* masnext,int Sx, int Sy, int Sz,double dx, double dy, double dz,double x0,double y0,double z0,double dt,double ddx,double ddy,double ddz,double *f)
{
	kernelMatrix<<<Sy-2,Sz-2>>>(masprev,masnext,Sx,Sy,Sz,dx,dy,dz,x0,y0,z0,dt,ddx,ddy,ddz,f);
	hipDeviceSynchronize();
}



